#include "hip/hip_runtime.h"
#include "defs.h"
#include "cuda_defs.h"

#define DTMIN 1e-8
__global__ void boundary_kernel(real *cons, real *intenergy, real *x1, real *x2, real *x3,
		int nx1, int nx2, int nx3, int size_x1, int size_x12, int nf, int ntot, int offset, real g, real time);

__global__ void nancheck_kernel(real *cons, int *out, int ntot,int nf);




real set_bc_timestep(real dt_max,
        real *d_cons,
        real *d_intenergy,
        real *d_dx1,
        real *d_dx2,
        real *d_dx3,
        real *d_x1,
        real *d_x2,
        real *d_x3,
        real *dt_arr,
        int *nan_arr,
        int *nan_res,
        GridCons *grid, Parameters *params);

__global__ void zero_flux_array(real *F1, real *F2, real *F3, int ntot, int nf) {
	for(int indx = blockIdx.x*blockDim.x + threadIdx.x; indx<ntot; indx+=blockDim.x*gridDim.x) {
		for(int n=0;n<nf;n++) {
			F1[indx + n*ntot] = 0.;
			F2[indx + n*ntot] = 0.;
			F3[indx + n*ntot] = 0.;
		}
	}
	return;
}
void algogas_single(real dt,
        real *d_cons,
        real *d_intenergy,
        real *d_UL_1,
        real *d_UR_1,
        real *d_F_1,
        real *d_UL_2,
        real *d_UR_2,
        real *d_F_2,
        real *d_UL_3,
        real *d_UR_3,
        real *d_F_3,
        real *d_dhalf,
        real *d_dx1,
        real *d_dx2,
        real *d_dx3,
        real *d_x1,
        real *d_x2,
        real *d_x3,
        real *dt_arr,
        int blocks,
        int threads,
        GridCons *grid, Parameters *params) {


    int nx1 = grid->nx[0];
    int nx2 = grid->nx[1];
    int nx3 = grid->nx[2];
    int ntot = grid->ntot;
    int size_x1 = grid->size_x1;
    int size_x12 = grid->size_x12;
    int nf = grid->nf;
    int offset = grid->offset;

   /* Add in operator split effects here. */


#ifdef CONDUCTION
    /* Add conduction */
     conduction_flux<<<grid->gridSize_conduction_flux, grid->blockSize_conduction_flux>>>(d_cons,
            d_intenergy,
            d_F_1,
            d_F_2,
            d_F_3,
            d_dx1,
            d_dx2,
            d_dx3,
            d_x1,
            d_x2,
            d_x3,
            params->gamma,
            nx1,
            nx2,
            nx3,
            size_x1,
            size_x12,
            ntot,
            offset,
            nf);
    cudaCheckError();

#endif

#ifdef VISCOSITY
    /* Add viscosity */
    /* Store velocities and divergence in one of
     * the reconstruction arrays
     */
     compute_divergence<<<grid->gridSize_divergence, grid->blockSize_divergence>>>(d_cons,
            d_UL_1,
            d_dx1,
            d_dx2,
            d_dx3,
            d_x1,
            d_x2,
            d_x3,
            nx1,
            nx2,
            nx3,
            size_x1,
            size_x12,
            ntot,
            offset,
            nf);
    cudaCheckError();
    viscous_flux<<<grid->gridSize_viscous_flux, grid->blockSize_viscous_flux>>>(d_UL_1,
    		d_cons,
           d_F_1,
           d_F_2,
           d_F_3,
           d_dx1,
           d_dx2,
           d_dx3,
           d_x1,
           d_x2,
           d_x3,
           nx1,
           nx2,
           nx3,
           size_x1,
           size_x12,
           ntot,
           offset,
           nf);
   cudaCheckError();
#endif
#if defined(CONDUCTION) || defined(VISCOSITY)
   /* Update conservative variables with diffusive fluxes */
	update_cons<<<grid->gridSize_update_cons, grid->blockSize_update_cons>>>(d_cons,
		   d_intenergy,
		   d_F_1,
		   d_F_2,
		   d_F_3,
		   d_dx1,
		   d_dx2,
		   d_dx3,
		   dt,
		   nx1,
		   nx2,
		   nx3,
		   size_x1,
		   size_x12,
		   ntot,
		   offset,
		   nf);
	cudaCheckError();
#endif




    /* X1 reconstruction */
	plm<<<grid->gridSize_plm, grid->blockSize_plm>>>(d_cons ,
		d_UL_1,
		d_UR_1,
		d_dx1,
		1,
		nx1,
		nx2,
		nx3,
		size_x1,
		size_x12,
		nf,
		ntot,
		offset,
		params->gamma-1,
		dt);
	cudaCheckError();
#ifdef POTENTIAL
	source_terms<<<grid->gridSize_source, grid->blockSize_source>>>(d_UL_1 ,
		d_UR_1,
		d_dx1,
		d_x1,
		d_x2,
		d_x3,
		1,
		nx1,
		nx2,
		nx3,
		size_x1,
		size_x12,
		nf,
		ntot,
		offset,
		params->gamma-1,
		dt);
	cudaCheckError();

#endif
	riemann_fluxes<<<grid->gridSize_riemann, grid->blockSize_riemann>>>(d_UL_1 ,
			d_UR_1 ,
			d_F_1 ,
			1,
			nx1,
			nx2,
			nx3,
			size_x1,
			size_x12,
			nf,
			ntot,
			offset,
			params->gamma);
	cudaCheckError();

	/* x2 reconstruction */
#ifdef DIMS2
    plm<<<grid->gridSize_plm, grid->blockSize_plm>>>(d_cons ,
            d_UL_2,
            d_UR_2,
            d_dx2,
            2,
            nx1,
            nx2,
            nx3,
            size_x1,
            size_x12,
            nf,
            ntot,
            offset,
            params->gamma-1,
            dt);
    cudaCheckError();
#ifdef POTENTIAL
	source_terms<<<grid->gridSize_source, grid->blockSize_source>>>(d_UL_2 ,
		d_UR_2,
		d_dx2,
		d_x1,
		d_x2,
		d_x3,
		2,
		nx1,
		nx2,
		nx3,
		size_x1,
		size_x12,
		nf,
		ntot,
		offset,
		params->gamma-1,
		dt);
	cudaCheckError();
    
#endif
	riemann_fluxes<<<grid->gridSize_riemann, grid->blockSize_riemann>>>(d_UL_2 ,
			d_UR_2 ,
			d_F_2 ,
			2,
			nx1,
			nx2,
			nx3,
			size_x1,
			size_x12,
			nf,
			ntot,
			offset,
			params->gamma);
	cudaCheckError();
#endif
#ifdef DIMS3
    plm<<<grid->gridSize_plm, grid->blockSize_plm>>>(d_cons ,
            d_UL_3,
            d_UR_3,
            d_dx3,
            3,
            nx1,
            nx2,
            nx3,
            size_x1,
            size_x12,
            nf,
            ntot,
            offset,
            params->gamma-1,
            dt);
    cudaCheckError();
#ifdef POTENTIAL
	source_terms<<<grid->gridSize_source, grid->blockSize_source>>>(d_UL_3 ,
		d_UR_3,
		d_dx3,
		d_x1,
		d_x2,
		d_x3,
		3,
		nx1,
		nx2,
		nx3,
		size_x1,
		size_x12,
		nf,
		ntot,
		offset,
		params->gamma-1,
		dt);
	cudaCheckError();

#endif
	riemann_fluxes<<<grid->gridSize_riemann, grid->blockSize_riemann>>>(d_UL_3 ,
			d_UR_3 ,
			d_F_3 ,
			3,
			nx1,
			nx2,
			nx3,
			size_x1,
			size_x12,
			nf,
			ntot,
			offset,
			params->gamma);
	cudaCheckError();
#endif


    /* Evolve interface states with transverse fluxes */

#ifdef CTU
#ifdef DIMS2
	transverse_update<<<grid->gridSize_transverse, grid->blockSize_transverse>>>(d_UL_1,
			d_UL_2,
			d_UL_3,
			d_UR_1,
			d_UR_2,
			d_UR_3,
			d_F_1 ,
			d_F_2 ,
			d_F_3 ,
			d_dx1,
			d_dx2,
			d_dx3,
			dt,
			nx1,
			nx2,
			nx3,
			size_x1,
			size_x12,
			ntot,
			offset,
			nf);
	cudaCheckError();
#ifdef POTENTIAL
	source_transverse_update<<<grid->gridSize_source_transverse, grid->blockSize_source_transverse>>>(d_cons,
			d_UL_1,
			d_UL_2,
			d_UL_3,
			d_UR_1,
			d_UR_2,
			d_UR_3,
			d_F_1 ,
			d_F_2 ,
			d_F_3 ,
			d_dx1,
			d_dx2,
			d_dx3,
			d_x1,
			d_x2,
			d_x3,
			dt,
			nx1,
			nx2,
			nx3,
			size_x1,
			size_x12,
			ntot,
			offset,
			nf);
	cudaCheckError();
#endif
    /* Compute new fluxes */
	riemann_fluxes<<<grid->gridSize_riemann, grid->blockSize_riemann>>>(d_UL_1 ,
			d_UR_1 ,
			d_F_1 ,
			1,
			nx1,
			nx2,
			nx3,
			size_x1,
			size_x12,
			nf,
			ntot,
			offset,
			params->gamma);
	cudaCheckError();
	riemann_fluxes<<<grid->gridSize_riemann, grid->blockSize_riemann>>>(d_UL_2 ,
			d_UR_2 ,
			d_F_2 ,
			2,
			nx1,
			nx2,
			nx3,
			size_x1,
			size_x12,
			nf,
			ntot,
			offset,
			params->gamma);
	cudaCheckError();
#ifdef DIMS3
	riemann_fluxes<<<grid->gridSize_riemann, grid->blockSize_riemann>>>(d_UL_3 ,
				d_UR_3 ,
				d_F_3 ,
				3,
				nx1,
				nx2,
				nx3,
				size_x1,
				size_x12,
				nf,
				ntot,
				offset,
				params->gamma);
		cudaCheckError();
#endif // DIMS3
#endif // DIMS2
#endif // CTU
#ifdef POTENTIAL
	compute_dhalf<<<grid->gridSize_update_cons, grid->blockSize_update_cons>>>(d_cons,
			d_dhalf,
			d_F_1,
			d_F_2,
			d_F_3,
			d_dx1,
			d_dx2,
			d_dx3,
			dt,
			nx1,
			nx2,
			nx3,
			size_x1,
			size_x12,
			ntot,
			offset,
			nf);
	cudaCheckError();

	update_source<<<grid->gridSize_update_source, grid->blockSize_update_source>>>(d_cons,
			d_dhalf,
			d_F_1,
			d_F_2,
			d_F_3,
			d_dx1,
			d_dx2,
			d_dx3,
			d_x1,
			d_x2,
			d_x3,
			nx1,
			nx2,
			nx3,
			size_x1,
			size_x12,
			nf,
			ntot,
			offset,dt);
	cudaCheckError();
    
#endif
    /* Final update */
    update_cons<<<grid->gridSize_update_cons, grid->blockSize_update_cons>>>(d_cons,
            d_intenergy,
            d_F_1,
            d_F_2,
            d_F_3,
            d_dx1,
            d_dx2,
            d_dx3,
            dt,
            nx1,
            nx2,
            nx3,
            size_x1,
            size_x12,
            ntot,
            offset,
            nf);
    cudaCheckError();


    return;
}

real algogas_dt(real dt, real dtout, int threads, int blocks, GridCons *grid, Parameters *params) {
    real end_time = grid->time + dtout;
    real dt_max;
    real history_dt = grid->time / (real)params->hout;

    int ntot = grid->ntot;
    int size_x1 = grid->size_x1;
    int size_x2 = grid->size_x2;
    int size_x3 = grid->size_x3;
    int size_x12 = grid->size_x12;
    int nf = grid->nf;
    int offset = grid->offset;

    int nan_res;
    real *d_cons, *d_intenergy;
    real *d_F_1, *d_UL_1, *d_UR_1;
    real *d_F_2, *d_UL_2, *d_UR_2;
    real *d_F_3, *d_UL_3, *d_UR_3;
    real *d_dx1, *d_dx2, *d_dx3;
    real *d_x1, *d_x2, *d_x3;
    real *dt_arr;
    real *d_dhalf;
    int *nan_arr;
    
    
    hipMalloc((void**)&d_dx1,sizeof(real)*size_x1);
	cudaCheckError();
	hipMemcpy(d_dx1,&grid->dx1[-NGHX1],sizeof(real)*size_x1,hipMemcpyHostToDevice);
	cudaCheckError();

	hipMalloc((void**)&d_dx2,sizeof(real)*size_x2);
	cudaCheckError();
	hipMemcpy(d_dx2,&grid->dx2[-NGHX2],sizeof(real)*size_x2,hipMemcpyHostToDevice);
	cudaCheckError();

	hipMalloc((void**)&d_dx3,sizeof(real)*size_x3);
	cudaCheckError();
	hipMemcpy(d_dx3,&grid->dx3[-NGHX3],sizeof(real)*size_x3,hipMemcpyHostToDevice);
	cudaCheckError();

	hipMalloc((void**)&d_x1,sizeof(real)*size_x1);
	cudaCheckError();
	hipMemcpy(d_x1,&grid->xc1[-NGHX1],sizeof(real)*size_x1,hipMemcpyHostToDevice);
	cudaCheckError();

	hipMalloc((void**)&d_x2,sizeof(real)*size_x2);
	cudaCheckError();
	hipMemcpy(d_x2,&grid->xc2[-NGHX2],sizeof(real)*size_x2,hipMemcpyHostToDevice);
	cudaCheckError();
	hipMalloc((void**)&d_x3,sizeof(real)*size_x3);
	cudaCheckError();
	hipMemcpy(d_x3,&grid->xc3[-NGHX3],sizeof(real)*size_x3,hipMemcpyHostToDevice);
	cudaCheckError();

	hipMalloc((void**)&d_cons,sizeof(real)*ntot*nf);
	cudaCheckError();
	hipMemcpy(d_cons,&grid->cons[-offset],sizeof(real)*ntot*nf,hipMemcpyHostToDevice);
	cudaCheckError();

	hipMalloc((void**)&d_intenergy,sizeof(real)*ntot);
	cudaCheckError();
	hipMemcpy(d_intenergy,&grid->intenergy[-offset],sizeof(real)*ntot,hipMemcpyHostToDevice);
	cudaCheckError();

	hipMalloc((void**)&d_UL_1,sizeof(real)*ntot*nf);
	cudaCheckError();

	hipMalloc((void**)&d_UR_1,sizeof(real)*ntot*nf);
	cudaCheckError();

	hipMalloc((void**)&d_F_1,sizeof(real)*ntot*nf);
	cudaCheckError();

	hipMalloc((void**)&d_UL_2,sizeof(real)*ntot*nf);
	cudaCheckError();

	hipMalloc((void**)&d_UR_2,sizeof(real)*ntot*nf);
	cudaCheckError();

	hipMalloc((void**)&d_F_2,sizeof(real)*ntot*nf);
	cudaCheckError();

	hipMalloc((void**)&d_UL_3,sizeof(real)*ntot*nf);
	cudaCheckError();

	hipMalloc((void**)&d_UR_3,sizeof(real)*ntot*nf);
	cudaCheckError();

	hipMalloc((void**)&d_F_3,sizeof(real)*ntot*nf);
	cudaCheckError();

	hipMalloc((void**)&d_dhalf,sizeof(real)*ntot);
	cudaCheckError();


	hipMalloc((void**)&dt_arr,sizeof(real)*(grid->gridSize_reduc));
	cudaCheckError();


	hipMalloc((void**)&nan_arr,sizeof(int)*(grid->gridSize_reduc));
	cudaCheckError();



   
    
    while (grid->time < end_time) { 
    	/* Zero flux arrays */
    	zero_flux_array<<<grid->gridSize_update_cons, grid->blockSize_update_cons>>>(d_F_1,d_F_2,d_F_3,ntot,nf);
    	cudaCheckError();
        /* Advance by dt */
        algogas_single(dt, 
            d_cons,
            d_intenergy,
            d_UL_1,
            d_UR_1,
            d_F_1,
            d_UL_2,
            d_UR_2,
            d_F_2,
            d_UL_3,
            d_UR_3,
		 	d_F_3,
            d_dhalf,
            d_dx1 + NGHX1,
            d_dx2 + NGHX2,
            d_dx3 + NGHX3,
            d_x1  + NGHX1,
            d_x2  + NGHX2,
            d_x3  + NGHX3,
            dt_arr,
            blocks,
            threads,
            grid, params);
    /* Set new timestep and bcs */
        grid->time += dt;
        dt_max = end_time - grid->time;
        dt = set_bc_timestep(dt_max, 
                d_cons,
                d_intenergy,
                d_dx1 + NGHX1,
                d_dx2 + NGHX2,
                d_dx3 + NGHX3,
                d_x1  + NGHX1,
                d_x2  + NGHX2,
                d_x3  + NGHX3,
                dt_arr,
                nan_arr,
                &nan_res,
                grid,params);
//        if (grid->time % history_dt == 0) {
//        	volume_averages(d_cons,d_intnergy,)
//        }
        if (nan_res) {
        	break;
        }

    }

    /* Convert to prim */

    cons_to_prim<<<grid->gridSize_update_cons, grid->blockSize_update_cons>>>(d_cons,d_intenergy,d_UL_1,params->gamma-1,
    	 grid->nx[0],grid->nx[1],grid->nx[2], size_x1, size_x12, ntot, offset, nf);
    cudaCheckError();

    /* Copy to host */
	hipMemcpy(&grid->cons[-offset],d_cons,sizeof(real)*ntot*nf,hipMemcpyDeviceToHost);
    cudaCheckError();

    hipMemcpy(&grid->prim[-offset],d_UL_1,sizeof(real)*ntot*nf,hipMemcpyDeviceToHost);
    cudaCheckError();

	hipMemcpy(&grid->intenergy[-offset],d_intenergy,sizeof(real)*ntot,hipMemcpyDeviceToHost);
    cudaCheckError();

    /* Free device arrays */
    hipFree(d_cons); cudaCheckError();
    hipFree(d_intenergy); cudaCheckError();
    hipFree(d_F_1); cudaCheckError();
    hipFree(d_UL_1); cudaCheckError();
    hipFree(d_UR_1); cudaCheckError();
    hipFree(d_F_2); cudaCheckError();
    hipFree(d_UL_2); cudaCheckError();
    hipFree(d_UR_2); cudaCheckError();
    hipFree(d_F_3); cudaCheckError();
    hipFree(d_UL_3); cudaCheckError();
    hipFree(d_UR_3); cudaCheckError();
    hipFree(d_dx1); cudaCheckError();
    hipFree(d_dx2); cudaCheckError();
    hipFree(d_dx3); cudaCheckError();
    hipFree(d_x1); cudaCheckError();
    hipFree(d_x2); cudaCheckError();
    hipFree(d_x3); cudaCheckError();
    hipFree(dt_arr); cudaCheckError();
    hipFree(d_dhalf); cudaCheckError();
    hipFree(nan_arr); cudaCheckError();

    return nan_res ? -dt : dt;
}
real algogas_firststep(real dtout, int threads, int blocks, int restart, int nostep, GridCons *grid, Parameters *params) {
    real end_time = grid->time + dtout;
    real dt_max,dt;
	int ntot = grid->ntot;
	int size_x1 = grid->size_x1;
	int size_x2 = grid->size_x2;
	int size_x3 = grid->size_x3;
	int nf = grid->nf;
	int offset = grid->offset;

	int nan_res;
	real *d_cons, *d_intenergy;
	real *d_F_1, *d_UL_1, *d_UR_1;
	real *d_F_2, *d_UL_2, *d_UR_2;
	real *d_F_3, *d_UL_3, *d_UR_3;
	real *d_dx1, *d_dx2, *d_dx3;
	real *d_x1, *d_x2, *d_x3;
	real *dt_arr;
	real *d_dhalf;
	int *nan_arr;


    hipMalloc((void**)&d_dx1,sizeof(real)*size_x1);
   	cudaCheckError();
   	hipMemcpy(d_dx1,&grid->dx1[-NGHX1],sizeof(real)*size_x1,hipMemcpyHostToDevice);
   	cudaCheckError();

   	hipMalloc((void**)&d_dx2,sizeof(real)*size_x2);
   	cudaCheckError();
   	hipMemcpy(d_dx2,&grid->dx2[-NGHX2],sizeof(real)*size_x2,hipMemcpyHostToDevice);
   	cudaCheckError();

   	hipMalloc((void**)&d_dx3,sizeof(real)*size_x3);
   	cudaCheckError();
   	hipMemcpy(d_dx3,&grid->dx3[-NGHX3],sizeof(real)*size_x3,hipMemcpyHostToDevice);
   	cudaCheckError();

   	hipMalloc((void**)&d_x1,sizeof(real)*size_x1);
   	cudaCheckError();
   	hipMemcpy(d_x1,&grid->xc1[-NGHX1],sizeof(real)*size_x1,hipMemcpyHostToDevice);
   	cudaCheckError();

   	hipMalloc((void**)&d_x2,sizeof(real)*size_x2);
   	cudaCheckError();
   	hipMemcpy(d_x2,&grid->xc2[-NGHX2],sizeof(real)*size_x2,hipMemcpyHostToDevice);
   	cudaCheckError();
   	hipMalloc((void**)&d_x3,sizeof(real)*size_x3);
   	cudaCheckError();
   	hipMemcpy(d_x3,&grid->xc3[-NGHX3],sizeof(real)*size_x3,hipMemcpyHostToDevice);
   	cudaCheckError();

   	hipMalloc((void**)&d_cons,sizeof(real)*ntot*nf);
   	cudaCheckError();
   	hipMemcpy(d_cons,&grid->cons[-offset],sizeof(real)*ntot*nf,hipMemcpyHostToDevice);
   	cudaCheckError();

   	hipMalloc((void**)&d_intenergy,sizeof(real)*ntot);
   	cudaCheckError();
   	hipMemcpy(d_intenergy,&grid->intenergy[-offset],sizeof(real)*ntot,hipMemcpyHostToDevice);
   	cudaCheckError();

   	hipMalloc((void**)&d_UL_1,sizeof(real)*ntot*nf);
   	cudaCheckError();

   	hipMalloc((void**)&d_UR_1,sizeof(real)*ntot*nf);
   	cudaCheckError();

   	hipMalloc((void**)&d_F_1,sizeof(real)*ntot*nf);
   	cudaCheckError();

   	hipMalloc((void**)&d_UL_2,sizeof(real)*ntot*nf);
   	cudaCheckError();

   	hipMalloc((void**)&d_UR_2,sizeof(real)*ntot*nf);
   	cudaCheckError();

   	hipMalloc((void**)&d_F_2,sizeof(real)*ntot*nf);
   	cudaCheckError();

   	hipMalloc((void**)&d_UL_3,sizeof(real)*ntot*nf);
   	cudaCheckError();

   	hipMalloc((void**)&d_UR_3,sizeof(real)*ntot*nf);
   	cudaCheckError();

   	hipMalloc((void**)&d_F_3,sizeof(real)*ntot*nf);
   	cudaCheckError();

   	hipMalloc((void**)&d_dhalf,sizeof(real)*ntot);
   	cudaCheckError();


   	hipMalloc((void**)&dt_arr,sizeof(real)*(grid->gridSize_reduc));
   	cudaCheckError();
   	hipMalloc((void**)&nan_arr,sizeof(int)*(grid->gridSize_reduc));
   	cudaCheckError();


	/* Zero flux arrays */
   	printf("THREADS X BLOCKS = %d x %d\n",threads,blocks);
   	zero_flux_array<<<grid->gridSize_update_cons, grid->blockSize_update_cons>>>(d_F_1,d_F_2,d_F_3,ntot,nf);
   	cudaCheckError();

    dt_max = end_time - grid->time;

    /* Set new timestep and bcs */

    dt = set_bc_timestep(dt_max, 
        d_cons,
        d_intenergy,
        d_dx1 + NGHX1,
        d_dx2 + NGHX2,
        d_dx3 + NGHX3,
        d_x1  + NGHX1,
        d_x2  + NGHX2,
        d_x3  + NGHX3,
        dt_arr,
        nan_arr,
        &nan_res,
        grid,params);

    /* Take one step */
    if (!nostep) {
		algogas_single(dt,
			d_cons,
			d_intenergy,
			d_UL_1,
			d_UR_1,
			d_F_1,
			d_UL_2,
			d_UR_2,
			d_F_2,
			d_UL_3,
			d_UR_3,
			d_F_3,
			d_dhalf,
			d_dx1 + NGHX1,
			d_dx2 + NGHX2,
			d_dx3 + NGHX3,
			d_x1 + NGHX1,
			d_x2 + NGHX2,
			d_x3 + NGHX3,
			dt_arr,
			blocks,
			threads,
			grid, params);

		grid->time += dt;
		/* Get new timestep */
		dt_max = end_time - grid->time;

		dt = set_bc_timestep(dt_max,
			d_cons,
			d_intenergy,
			d_dx1 + NGHX1,
			d_dx2 + NGHX2,
			d_dx3 + NGHX3,
			d_x1  + NGHX1,
			d_x2  + NGHX2,
			d_x3  + NGHX3,
			dt_arr,
	        nan_arr,
	        &nan_res,
			grid,params);
    }
    else {
    	grid->time += dt;
    }
    /* Copy results to host */
    cons_to_prim<<<grid->gridSize_update_cons, grid->blockSize_update_cons>>>(d_cons,d_intenergy,d_UL_1,params->gamma-1,
    	 grid->nx[0],grid->nx[1],grid->nx[2], size_x1, grid->size_x12, ntot, offset, nf);
    cudaCheckError();


	hipMemcpy(&grid->cons[-offset],d_cons,sizeof(real)*ntot*nf,hipMemcpyDeviceToHost);
    cudaCheckError();

    hipMemcpy(&grid->prim[-offset],d_UL_1,sizeof(real)*ntot*nf,hipMemcpyDeviceToHost);
    cudaCheckError();


	hipMemcpy(&grid->intenergy[-offset],d_intenergy,sizeof(real)*ntot,hipMemcpyDeviceToHost);
    cudaCheckError();


    /* Free device arrays */
    hipFree(d_cons); cudaCheckError();
    hipFree(d_intenergy);cudaCheckError();
    hipFree(d_F_1); cudaCheckError();
    hipFree(d_UL_1); cudaCheckError();
    hipFree(d_UR_1); cudaCheckError();
    hipFree(d_F_2); cudaCheckError();
    hipFree(d_UL_2); cudaCheckError();
    hipFree(d_UR_2); cudaCheckError();
    hipFree(d_F_3); cudaCheckError();
    hipFree(d_UL_3); cudaCheckError();
    hipFree(d_UR_3); cudaCheckError();
    hipFree(d_dx1); cudaCheckError();
    hipFree(d_dx2); cudaCheckError();
    hipFree(d_dx3); cudaCheckError();
    hipFree(d_x1); cudaCheckError();
    hipFree(d_x2); cudaCheckError();
    hipFree(d_x3); cudaCheckError();
    hipFree(dt_arr); cudaCheckError();
    hipFree(d_dhalf); cudaCheckError();
    hipFree(nan_arr); cudaCheckError();


    return nan_res ? -dt : dt;
}
//}


real set_bc_timestep(real dt_max, 
        real *d_cons,
        real *d_intenergy,
        real *d_dx1,
        real *d_dx2,
        real *d_dx3,
        real *d_x1,
        real *d_x2,
        real *d_x3,
        real *dt_arr,
        int *nan_arr,
        int *nan_res,
        GridCons *grid, Parameters *params) {


    int nx1 = grid->nx[0];
    int nx2 = grid->nx[1];
    int nx3 = grid->nx[2];
    int ntot = grid->ntot;
    int size_x1 = grid->size_x1;
    int size_x12 = grid->size_x12;
    int nf = grid->nf;
    int offset = grid->offset;
    real dt;
    real h_dt_arr[1024];
    int h_nan_arr[1024];
    int blocks = grid->gridSize_reduc;
//    hipMemcpy(&grid->cons[-offset],d_cons,sizeof(real)*ntot*nf,hipMemcpyDeviceToHost);
//	cudaCheckError();
//
//	hipMemcpy(&grid->intenergy[-offset],d_intenergy,sizeof(real)*ntot,hipMemcpyDeviceToHost);
//	cudaCheckError();

//	printf("%lg\n",dt);
    /* Calculate new timestep */

    timestep_kernel<<<grid->gridSize_reduc , grid->blockSize_reduc >>>(d_cons,
    		d_dx1,
    		d_dx2,
    		d_dx3,
    		d_x1,
    		d_x2,
    		d_x3,
    		dt_arr,
    		nx1,nx2,nx3,
    		size_x1,size_x12,
            ntot,offset,params->gamma);
    cudaCheckError();

    /* Do final reduction on host */
    hipMemcpy(h_dt_arr,dt_arr,sizeof(real)*blocks,hipMemcpyDeviceToHost);
    cudaCheckError();

    dt = FLOATMAX;
    for(int i=0;i<blocks;i++) {
    	if (h_dt_arr[i] < dt) dt = h_dt_arr[i];
    }
    dt *= params->cfl;
//    timestep_kernel_final<<<1,blocks>>>(dt_arr,dt_arr,blocks,params->cfl);
//    cudaCheckError();
//
//    hipMemcpy(&dt,dt_arr,sizeof(real),hipMemcpyDeviceToHost);
//    cudaCheckError();

//	curr_min = FLOATMAX;
//	real pres,dt1,cs;
//	for(int i=0;i<nx1;i++) {
//        pres = grid->intenergy[i] * params->gamma_1;
//
//        cs = sqrt( params->gamma* pres/grid->cons[i]);
//        dt1 = grid->dx1[i]/(fabs(grid->cons[i + 1*ntot]/grid->cons[i]) + cs);
//        //printf("%lg\t%lg\t%lg\n",cs,dt1,curr_min);
//        if (dt1 < curr_min) curr_min = dt1;
//	}

    if (dt < DTMIN){
        printf("Timestep %.4e fell below minimum value of %.1e\n",dt,DTMIN);
        exit(0);
    }
    if (dt > dt_max) dt = dt_max;




    /* Set boundaries */

    boundary_kernel<<<grid->gridSize_update_cons, grid->blockSize_update_cons>>>(d_cons,d_intenergy,d_x1,d_x2,d_x3,nx1,nx2,nx3,size_x1,size_x12,nf,ntot,offset,params->gamma,grid->time);
    cudaCheckError();
    hipDeviceSynchronize();
    cudaCheckError();
    
    /* Check for NaN */
    nancheck_kernel<<<grid->gridSize_reduc , grid->blockSize_reduc >>>(d_cons, nan_arr, ntot,nf);
    cudaCheckError();

    /* Do final reduction on host */
    hipMemcpy(h_nan_arr,nan_arr,sizeof(int)*blocks,hipMemcpyDeviceToHost);
    cudaCheckError();

    *nan_res = FALSE;
    for(int i=0;i<blocks;i++) *nan_res |= h_nan_arr[i];

    return dt;
}
__global__ void boundary_kernel(real *cons, real *intenergy, real *x1, real *x2, real *x3,
		int nx1, int nx2, int nx3, int size_x1, int size_x12, int nf, int ntot, int offset, real g, real time) {

    int i,j,k,indxg;
    for(indxg = blockIdx.x*blockDim.x + threadIdx.x; indxg<ntot; indxg+=blockDim.x*gridDim.x) {
    	unpack_indices(indxg,&i,&j,&k,size_x1,size_x12);

        if ((i>=-NGHX1)&&(i<0)&&(j>=-NGHX2)&&(j<nx2+NGHX2)&&(k>=-NGHX3)&&(k<nx3+NGHX3)) {
        /* Lower x1 */
        	x1_boundary_inner(indxg,i,j,k,cons,intenergy,x1,x2,x3,nx1,nx2,nx3,ntot,nf,size_x1,size_x12,offset,g,time);
        }
        else if ((i>=nx1)&&(i<nx1+NGHX1)&&(j>=-NGHX2)&&(j<nx2+NGHX2)&&(k>=-NGHX3)&&(k<nx3+NGHX3))  {
         /* Upper x1 */
         	x1_boundary_outer(indxg,i,j,k,cons,intenergy,x1,x2,x3,nx1,nx2,nx3,ntot,nf,size_x1,size_x12,offset,g,time);
        }
#ifdef DIMS2
        else if ((j>=-NGHX2)&&(j<0)&&(i>=-NGHX1)&&(i<nx1+NGHX1)&&(k>=-NGHX3)&&(k<nx3+NGHX3)) {
        /* Lower x2 */
        	x2_boundary_inner(indxg,i,j,k,cons,intenergy,x1,x2,x3,nx1,nx2,nx3,ntot,nf,size_x1,size_x12,offset,g,time);
        }
        else if ((j>=nx2)&&(j<nx2+NGHX2)&&(i>=-NGHX1)&&(i<nx1+NGHX1)&&(k>=-NGHX3)&&(k<nx3+NGHX3)) {
        /* Upper x2 */
            x2_boundary_outer(indxg,i,j,k,cons,intenergy,x1,x2,x3,nx1,nx2,nx3,ntot,nf,size_x1,size_x12,offset,g,time);
        }
#endif
#ifdef DIMS3
        else if ((k>=-NGHX2)&&(k<0)&&(i>=-NGHX1)&&(i<nx1+NGHX1)&&(j>=-NGHX2)&&(j<nx2+NGHX2)) {
        /* Lower x3 */
        	x3_boundary_inner(indxg,i,j,k,cons,intenergy,x1,x2,x3,nx1,nx2,nx3,ntot,nf,size_x1,size_x12,offset,g,time);
        }
        else if ((k>=nx3)&&(k<nx3+NGHX3)&&(i>=-NGHX1)&&(i<nx1+NGHX1)&&(j>=-NGHX2)&&(j<nx2+NGHX2)) {
        /* Upper x3 */
            x3_boundary_outer(indxg,i,j,k,cons,intenergy,x1,x2,x3,nx1,nx2,nx3,ntot,nf,size_x1,size_x12,offset,g,time);
        }
#endif
    }
    return;
}

__global__ void nancheck_kernel(real *cons, int *out, int ntot,int nf) {
    int indx,n;

    int curr_res = FALSE;

    for(indx = blockIdx.x*blockDim.x + threadIdx.x; indx<ntot;indx +=blockDim.x*gridDim.x) {
    	for(n=0;n<nf;n++) curr_res |= (cons[indx +n*ntot] != cons[indx + n*ntot]);

    }
    curr_res = blockReduceBoolOR(curr_res);
    if (threadIdx.x ==0) out[blockIdx.x]=curr_res;
    return;
}

