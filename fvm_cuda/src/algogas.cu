#include "defs.h"
#include "cuda_defs.h"

#define DTMIN 1e-8

//extern "C" {
void algogas_single(real dt,
        real *d_cons,
        real *d_intenergy,
        real *d_UL_1,
        real *d_UR_1,
        real *d_F_1,
        real *d_UL_2,
        real *d_UR_2,
        real *d_F_2,
        real *d_dhalf,
        real *d_dx1,
        real *d_dx2,
        real *d_x1,
        real *d_x2,
        real *dt_arr,
        int blocks,
        int threads,
        GridCons *grid, Parameters *params) {


    int nx1 = grid->nx[0];
    int nx2 = grid->nx[1];
    int ntot = grid->ntot;
    int size_x1 = grid->size_x[0];
    int nf = grid->nf;
    int offset = grid->offset;

    
#ifdef CONDUCTION
     conduction_flux<<<blocks,threads>>>(d_cons,
            d_intenergy,
            d_F_1,
            d_F_2,
            d_dx1,
            d_dx2,
            d_x1,
            d_x2,
            params->gamma,
            dt,
            nx1,
            nx2,
            size_x1,
            ntot,
            offset,
            nf);
    cudaCheckError();
    conduction_update<<<blocks,threads>>>(d_cons,
            d_intenergy,
            d_F_1,
            d_F_2,
            d_dx1,
            d_dx2,
            dt,
            nx1,
            nx2,
            size_x1,
            ntot,
            offset,
            nf);
    cudaCheckError();
#endif




    if (nx1 > 1) {
        plm<<<blocks,threads>>>(d_cons ,
            d_UL_1, 
            d_UR_1,
            d_dx1,
            1,
            nx1,
            nx2,
            size_x1,
            nf,
            ntot,
            offset,
            params->gamma_1,
            dt);
        cudaCheckError();
#ifdef POTENTIAL
        source_terms<<<blocks,threads>>>(d_UL_1 ,
            d_UR_1,
            d_dx1,
            d_x1,
            d_x2,
            1,
            nx1,
            nx2,
            size_x1,
            nf,
            ntot,
            offset,
            params->gamma_1,
            dt);
        cudaCheckError();
    
#endif
        riemann_fluxes<<<blocks,threads>>>(d_UL_1 ,
                d_UR_1 ,
                d_F_1 ,
                1,
                nx1,
                nx2,
                size_x1,
                nf,
                ntot,
                offset,
                params->gamma);
        cudaCheckError();
    }
    if (nx2 > 1) {

    plm<<<blocks,threads>>>(d_cons ,
            d_UL_2,
            d_UR_2,
            d_dx2,
            2,
            nx1,
            nx2,
            size_x1,
            nf,
            ntot,
            offset,
            params->gamma_1,
            dt);
    cudaCheckError();
#ifdef POTENTIAL
        source_terms<<<blocks,threads>>>(d_UL_2 ,
            d_UR_2,
            d_dx2,
            d_x1,
            d_x2,
            2,
            nx1,
            nx2,
            size_x1,
            nf,
            ntot,
            offset,
            params->gamma_1,
            dt);
        cudaCheckError();
    
#endif
        riemann_fluxes<<<blocks,threads>>>(d_UL_2 ,
                d_UR_2 ,
                d_F_2 ,
                2,
                nx1,
                nx2,
                size_x1,
                nf,
                ntot,
                offset,
                params->gamma);
        cudaCheckError();
    }


    /* Evolve interface states with transverse fluxes */

#ifdef CTU
    if ((nx1 > 1)&&(nx2 > 1)) {
        transverse_update<<<blocks,threads>>>(d_UL_1,
                d_UL_2,
                d_UR_1,
                d_UR_2,
                d_F_1 ,
                d_F_2 ,
                d_dx1,
                d_dx2,
                dt,
                nx1,
                nx2,
                size_x1,
                ntot,
                offset,
                nf);
        cudaCheckError();
        source_transverse_update<<<blocks,threads>>>(d_cons,
                d_UL_1,
                d_UL_2,
                d_UR_1,
                d_UR_2,
                d_F_1 ,
                d_F_2 ,
                d_dx1,
                d_dx2,
                d_x1,
                d_x2,
                dt,
                nx1,
                nx2,
                size_x1,
                ntot,
                offset,
                nf);
        cudaCheckError();
    /* Compute new fluxes */
        riemann_fluxes<<<blocks,threads>>>(d_UL_1 , 
                d_UR_1 ,
                d_F_1 ,
                1,
                nx1,
                nx2,
                size_x1,
                nf,
                ntot,
                offset,
                params->gamma);
        cudaCheckError();
        riemann_fluxes<<<blocks,threads>>>(d_UL_2 ,
                d_UR_2 ,
                d_F_2 ,
                2,
                nx1,
                nx2,
                size_x1,
                nf,
                ntot,
                offset,
                params->gamma);
        cudaCheckError();
    }
#endif 
#ifdef POTENTIAL
    compute_dhalf<<<blocks,threads>>>(d_cons,
            d_dhalf,
            d_F_1,
            d_F_2,
            d_dx1,
            d_dx2,
            dt,
            nx1,
            nx2,
            size_x1,
            ntot,
            offset,
            nf);
    update_source<<<blocks,threads>>>(d_cons,
            d_dhalf,
            d_F_1,
            d_F_2,
            d_dx1,
            d_dx2,
            d_x1,
            d_x2,
            nx1,
            nx2,
            size_x1,
            nf,
            ntot,
            offset,dt);
        cudaCheckError();
    
#endif
    /* Final update */
    update_cons<<<blocks,threads>>>(d_cons,
            d_intenergy,
            d_F_1,
            d_F_2,
            d_dx1,
            d_dx2,
            dt,
            nx1,
            nx2,
            size_x1,
            ntot,
            offset,
            nf);
    cudaCheckError();

   
    




    return;
}

real algogas_dt(real dt, real dtout, int threads, int blocks, GridCons *grid, FluxCons *fluxes, Parameters *params) {
    real end_time = grid->time + dtout;
    real dt_max;
    int ntot = grid->ntot;
    int nf = grid->nf;
    int size_x1 = grid->size_x[0];
    int size_x2 = grid->size_x[1];
    int offset = grid->offset;


    real *d_cons, *d_intenergy;
    real *d_F_1, *d_UL_1, *d_UR_1;
    real *d_F_2, *d_UL_2, *d_UR_2;
    real *d_dx1, *d_dx2;
    real *d_x1, *d_x2;
    real *dt_arr;
    real *d_dhalf;
    
    
    hipMalloc((void**)&d_dx1,sizeof(real)*size_x1);
	cudaCheckError();
	hipMemcpy(d_dx1,&grid->dx1[-NGHX1],sizeof(real)*size_x1,hipMemcpyHostToDevice);
	cudaCheckError();

	hipMalloc((void**)&d_dx2,sizeof(real)*size_x2);
	cudaCheckError();
	hipMemcpy(d_dx2,&grid->dx2[-NGHX2],sizeof(real)*size_x2,hipMemcpyHostToDevice);
	cudaCheckError();

	hipMalloc((void**)&d_x1,sizeof(real)*size_x1);
	cudaCheckError();
	hipMemcpy(d_x1,&grid->xc1[-NGHX1],sizeof(real)*size_x1,hipMemcpyHostToDevice);
	cudaCheckError();

	hipMalloc((void**)&d_x2,sizeof(real)*size_x2);
	cudaCheckError();
	hipMemcpy(d_x2,&grid->xc2[-NGHX2],sizeof(real)*size_x2,hipMemcpyHostToDevice);
	cudaCheckError();

	hipMalloc((void**)&d_cons,sizeof(real)*ntot*nf);
	cudaCheckError();
	hipMemcpy(d_cons,&grid->cons[-offset],sizeof(real)*ntot*nf,hipMemcpyHostToDevice);
	cudaCheckError();

	hipMalloc((void**)&d_intenergy,sizeof(real)*ntot);
	cudaCheckError();
	hipMemcpy(d_intenergy,&grid->intenergy[-offset],sizeof(real)*ntot,hipMemcpyHostToDevice);
	cudaCheckError();

	hipMalloc((void**)&d_UL_1,sizeof(real)*ntot*nf);
	cudaCheckError();

	hipMalloc((void**)&d_UR_1,sizeof(real)*ntot*nf);
	cudaCheckError();

	hipMalloc((void**)&d_F_1,sizeof(real)*ntot*nf);
	cudaCheckError();

	hipMalloc((void**)&d_UL_2,sizeof(real)*ntot*nf);
	cudaCheckError();

	hipMalloc((void**)&d_UR_2,sizeof(real)*ntot*nf);
	cudaCheckError();

	hipMalloc((void**)&d_F_2,sizeof(real)*ntot*nf);
	cudaCheckError();

	hipMalloc((void**)&d_dhalf,sizeof(real)*ntot);
	cudaCheckError();


	hipMalloc((void**)&dt_arr,sizeof(real)*blocks);
	cudaCheckError();

   
    
    while (grid->time < end_time) { 
        /* Advance by dt */
        algogas_single(dt, 
            d_cons,
            d_intenergy,
            d_UL_1,
            d_UR_1,
            d_F_1,
            d_UL_2,
            d_UR_2,
            d_F_2,
            d_dhalf,
            d_dx1 + NGHX1,
            d_dx2 + NGHX2,
            d_x1  + NGHX1,
            d_x2  + NGHX2,
            dt_arr,
            blocks,
            threads,
            grid, params);
    /* Set new timestep and bcs */
        grid->time += dt;
        dt_max = end_time - grid->time;
        dt = set_bc_timestep(dt_max, 
                d_cons,
                d_intenergy,
                d_dx1 + NGHX1,
                d_dx2 + NGHX2,
                d_x1  + NGHX1,
                d_x2  + NGHX2,
                dt_arr,
                blocks,
                threads,
                grid,params);

    }

    /* Copy to host */
	hipMemcpy(&grid->cons[-offset],d_cons,sizeof(real)*ntot*nf,hipMemcpyDeviceToHost);
    cudaCheckError();

	hipMemcpy(&grid->intenergy[-offset],d_intenergy,sizeof(real)*ntot,hipMemcpyDeviceToHost);
    cudaCheckError();

    /* Free device arrays */
    hipFree(d_cons);
    hipFree(d_intenergy);
    hipFree(d_F_1);
    hipFree(d_UL_1);
    hipFree(d_UR_1);
    hipFree(d_F_2);
    hipFree(d_UL_2);
    hipFree(d_UR_2);
    hipFree(d_dx1);
    hipFree(d_dx2);
    hipFree(d_x1);
    hipFree(d_x2);
    hipFree(dt_arr);
    hipFree(d_dhalf);

    return dt;
}
real algogas_firststep(real dtout, int threads, int blocks, GridCons *grid, FluxCons *fluxes, Parameters *params) {
    real end_time = grid->time + dtout;
    real dt_max,dt;
    int ntot = grid->ntot;
    int nf = grid->nf;
    int size_x1 = grid->size_x[0];
    int size_x2 = grid->size_x[1];
    int offset = grid->offset;


    real *d_cons, *d_intenergy;
    real *d_F_1, *d_UL_1, *d_UR_1;
    real *d_F_2, *d_UL_2, *d_UR_2;
    real *d_dx1, *d_dx2;
    real *d_x1, *d_x2;
    real *dt_arr;
    real *d_dhalf;


    hipMalloc((void**)&d_dx1,sizeof(real)*size_x1);
	cudaCheckError();
	hipMemcpy(d_dx1,&grid->dx1[-NGHX1],sizeof(real)*size_x1,hipMemcpyHostToDevice);
	cudaCheckError();

	hipMalloc((void**)&d_dx2,sizeof(real)*size_x2);
	cudaCheckError();
	hipMemcpy(d_dx2,&grid->dx2[-NGHX2],sizeof(real)*size_x2,hipMemcpyHostToDevice);
	cudaCheckError();

	hipMalloc((void**)&d_x1,sizeof(real)*size_x1);
	cudaCheckError();
	hipMemcpy(d_x1,&grid->xc1[-NGHX1],sizeof(real)*size_x1,hipMemcpyHostToDevice);
	cudaCheckError();

	hipMalloc((void**)&d_x2,sizeof(real)*size_x2);
	cudaCheckError();
	hipMemcpy(d_x2,&grid->xc2[-NGHX2],sizeof(real)*size_x2,hipMemcpyHostToDevice);
	cudaCheckError();

	hipMalloc((void**)&d_cons,sizeof(real)*ntot*nf);
	cudaCheckError();
	hipMemcpy(d_cons,&grid->cons[-offset],sizeof(real)*ntot*nf,hipMemcpyHostToDevice);
	cudaCheckError();

	hipMalloc((void**)&d_intenergy,sizeof(real)*ntot);
	cudaCheckError();
	hipMemcpy(d_intenergy,&grid->intenergy[-offset],sizeof(real)*ntot,hipMemcpyHostToDevice);
	cudaCheckError();

	hipMalloc((void**)&d_UL_1,sizeof(real)*ntot*nf);
	cudaCheckError();

	hipMalloc((void**)&d_UR_1,sizeof(real)*ntot*nf);
	cudaCheckError();

	hipMalloc((void**)&d_F_1,sizeof(real)*ntot*nf);
	cudaCheckError();

	hipMalloc((void**)&d_UL_2,sizeof(real)*ntot*nf);
	cudaCheckError();

	hipMalloc((void**)&d_UR_2,sizeof(real)*ntot*nf);
	cudaCheckError();

	hipMalloc((void**)&d_F_2,sizeof(real)*ntot*nf);
	cudaCheckError();

	hipMalloc((void**)&d_dhalf,sizeof(real)*ntot);
	cudaCheckError();


	hipMalloc((void**)&dt_arr,sizeof(real)*blocks);
	cudaCheckError();

   

    dt_max = end_time - grid->time;

    /* Set new timestep and bcs */

    dt = set_bc_timestep(dt_max, 
        d_cons,
        d_intenergy,
        d_dx1 + NGHX1,
        d_dx2 + NGHX2,
        d_x1  + NGHX1,
        d_x2  + NGHX2,
        dt_arr,
        blocks,
        threads,
        grid,params);

    /* Take one step */

    algogas_single(dt,
        d_cons,
        d_intenergy,
        d_UL_1,
        d_UR_1,
        d_F_1,
        d_UL_2,
        d_UR_2,
        d_F_2,
        d_dhalf,
        d_dx1 + NGHX1,
        d_dx2 + NGHX2,
        d_x1 + NGHX1,
        d_x2 + NGHX2,
        dt_arr,
        blocks,
        threads,
        grid, params);

    grid->time += dt;
    /* Get new timestep */
    dt_max = end_time - grid->time;

    dt = set_bc_timestep(dt_max,
        d_cons,
        d_intenergy,
        d_dx1 + NGHX1,
        d_dx2 + NGHX2,
        d_x1  + NGHX1,
        d_x2  + NGHX2,
        dt_arr,
        blocks,
        threads,
        grid,params);
    
    /* Copy results to host */
	hipMemcpy(&grid->cons[-offset],d_cons,sizeof(real)*ntot*nf,hipMemcpyDeviceToHost);
    cudaCheckError();

	hipMemcpy(&grid->intenergy[-offset],d_intenergy,sizeof(real)*ntot,hipMemcpyDeviceToHost);
    cudaCheckError();

    /* Free device arrays */
    hipFree(d_cons); cudaCheckError();
    hipFree(d_intenergy);cudaCheckError();
    hipFree(d_F_1); cudaCheckError();
    hipFree(d_UL_1); cudaCheckError();
    hipFree(d_UR_1); cudaCheckError();
    hipFree(d_F_2); cudaCheckError();
    hipFree(d_UL_2); cudaCheckError();
    hipFree(d_UR_2); cudaCheckError();
    hipFree(d_dx1); cudaCheckError();
    hipFree(d_dx2); cudaCheckError();
    hipFree(d_x1); cudaCheckError();
    hipFree(d_x2); cudaCheckError();
    hipFree(dt_arr); cudaCheckError();
    hipFree(d_dhalf); cudaCheckError();

    return dt;
}
//}


real set_bc_timestep(real dt_max, 
        real *d_cons,
        real *d_intenergy,
        real *d_dx1,
        real *d_dx2,
        real *d_x1,
        real *d_x2,
        real *dt_arr,
        int blocks,
        int threads,
        GridCons *grid, Parameters *params) {


    int nx1 = grid->nx[0];
    int nx2 = grid->nx[1];
    int ntot = grid->ntot;
    int size_x1 = grid->size_x[0];
    int nf = grid->nf;
    int offset = grid->offset;
    real dt;

    /* Calculate new timestep */
    timestep_kernel<<<blocks,threads>>>(d_cons,d_dx1,d_dx2,d_x1,d_x2,dt_arr,nx1,nx2,size_x1,
            ntot,offset,params->gamma,params->gamma_1);
    cudaCheckError();
    
    timestep_kernel_final<<<1,blocks>>>(dt_arr,dt_arr,blocks,params->cfl);
    cudaCheckError();

    hipMemcpy(&dt,dt_arr,sizeof(real),hipMemcpyDeviceToHost);
    cudaCheckError();
   
    if (dt < DTMIN){
        printf("Timestep %.4e fell below minimum value of %.1e\n",dt,DTMIN);
        //exit(1);
    }
    if (dt > dt_max) dt = dt_max;



    /* Set boundaries */

    boundary_kernel<<<blocks,threads>>>(d_cons,d_intenergy,d_x1,d_x2,nx1,nx2,size_x1,nf,ntot,offset,params->gamma,grid->time);
    cudaCheckError();
    
    return dt;
}
